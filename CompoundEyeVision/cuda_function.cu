#include "hip/hip_runtime.h"
#include "cuda_function.h"


__global__ void map_kernel(cuda::PtrStepSz<double> map_Hinv, cuda::PtrStepSz<float> mapx, cuda::PtrStepSz<float> mapy)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    mapx(j, i) = ((map_Hinv(0, 0) * i + map_Hinv(0, 1) * j + map_Hinv(0, 2)) / (map_Hinv(2, 0) * i + map_Hinv(2, 1) * j + map_Hinv(2, 2)));
    mapy(j, i) = ((map_Hinv(1, 0) * i + map_Hinv(1, 1) * j + map_Hinv(1, 2)) / (map_Hinv(2, 0) * i + map_Hinv(2, 1) * j + map_Hinv(2, 2)));
}

/**
 * @brief stitch_kernel
 * 在合并两个图层时，以非零像素为准。如果顶层图像的像素值为零，则选择底层图像的像素值。
 * 这种操作通常用于遮罩效果，其中顶层图像的非零像素表示前景，零像素表示透明区域，而底层图像则充当背景。
 * @param bottom
 * @param top
 * @param dst
 */
__global__ void stitch_kernel(cuda::PtrStepSz<uchar3> bottom, cuda::PtrStepSz<uchar3> top, cuda::PtrStepSz<uchar3> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    dst(j, i).x = top(j, i).x > 0 ? top(j, i).x : bottom(j, i).x;
    dst(j, i).y = top(j, i).y > 0 ? top(j, i).y : bottom(j, i).y;
    dst(j, i).z = top(j, i).z > 0 ? top(j, i).z : bottom(j, i).z;
}

/**
 * @brief stitchOpt_kernel
 * 通过线程坐标 (i, j) 计算每个像素位置。top_buffer 中的像素值递减，
 * 且如果 diff_val 大于阈值（20），则 top_buffer 中的像素值被设置为 30。
 * 根据 top_buffer 的值，决定选择哪个顶层或底层的像素值作为输出。
 * @param bottom
 * @param top1
 * @param top2
 * @param top_buffer
 * @param dst
 * @param odd
 */
__global__ void stitchOpt_kernel(cuda::PtrStepSz<uchar3> bottom, cuda::PtrStepSz<uchar3> top1, cuda::PtrStepSz<uchar3> top2,
                                 cuda::PtrStepSz<uchar> top_buffer, cuda::PtrStepSz<uchar3> dst, bool odd)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if(top_buffer(j, i) > 0) top_buffer(j, i)--;
    int diff_val = 0;

    if(odd)
        diff_val = abs( bottom(j, i).x - top1(j, i).x) + abs(bottom(j, i).y - top1(j, i).y) + abs(bottom(j, i).z - top1(j, i).z);
    else
        diff_val = abs( bottom(j, i).x - top2(j, i).x) + abs(bottom(j, i).y - top2(j, i).y) + abs(bottom(j, i).z - top2(j, i).z);
    if(diff_val > 20) top_buffer(j, i) = 30;

    if (top_buffer(j, i) > 0 ) {
        if(odd){
            dst(j, i).x = top1(j, i).x > 0 ? top1(j, i).x : bottom(j, i).x;
            dst(j, i).y = top1(j, i).y > 0 ? top1(j, i).y : bottom(j, i).y;
            dst(j, i).z = top1(j, i).z > 0 ? top1(j, i).z : bottom(j, i).z;
        }
        else {
            dst(j, i).x = top2(j, i).x > 0 ? top2(j, i).x : bottom(j, i).x;
            dst(j, i).y = top2(j, i).y > 0 ? top2(j, i).y : bottom(j, i).y;
            dst(j, i).z = top2(j, i).z > 0 ? top2(j, i).z : bottom(j, i).z;
        }
    }
    else
        return;
}

__global__ void darkch_kernel(cuda::PtrStepSz<uchar3> src, cuda::PtrStepSz<uchar> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    dst(j, i) = Min(src(j, i).x, Min(src(j, i).y, src(j, i).z));
}

__global__ void tran_kernel(cuda::PtrStepSz<uchar> src, cuda::PtrStepSz<uchar> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    uchar val = 255 - float( (src(j, i)) * 0.95);
    dst(j, i) = val;
}

__global__ void dehaze_kernal(cuda::PtrStepSz<uchar3> src, cuda::PtrStepSz<uchar3> dst, cuda::PtrStepSz<uchar> tran, float air_val0, float air_val1, float air_val2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    float r = float((float)tran(j, i) < 25 ? 25 : (float)tran(j, i)) / 255;

    dst(j, i).x = Min(float(src(j, i).x - float(255 - tran(j, i)) * (1 - air_val0)) / r, 255);
    dst(j, i).y = Min(float(src(j, i).y - float(255 - tran(j, i)) * (1 - air_val1)) / r, 255);
    dst(j, i).z = Min(float(src(j, i).z - float(255 - tran(j, i)) * (1 - air_val2)) / r, 255);
}

__global__ void enhanceTrans_kernel(cuda::PtrStepSz<uchar> src, cuda::PtrStepSz<uchar> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    uchar val = 255 - float((src(j, i)) * 0.8);
    dst(j, i) = (val > 204) ? val : 204;
}

__global__ void enhance_kernel(cuda::PtrStepSz<uchar3> src, cuda::PtrStepSz<uchar3> dst, cuda::PtrStepSz<uchar> tran, float k_val , float air_val0, float air_val1, float air_val2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    float tran_scale = ((float)tran(j, i) / 255);

    dst(j, i).x = Min((float)(((src(j, i).x) / tran_scale * k_val) * air_val0), 255);
    dst(j, i).y = Min((float)(((src(j, i).y) / tran_scale * k_val) * air_val1), 255);
    dst(j, i).z = Min((float)(((src(j, i).z) / tran_scale * k_val) * air_val2), 255);
}
__global__ void dehazeOpt_kernal(cuda::PtrStepSz<uchar3> frame1, cuda::PtrStepSz<uchar3> frame2, cuda::PtrStepSz<uchar3> result, cuda::PtrStepSz<uchar> dark,
                                 cuda::PtrStepSz<uchar> tran, cuda::PtrStepSz<uchar> buffer, bool odd, float air_val0, float air_val1, float air_val2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if(buffer(j, i) > 0) buffer(j, i)--;
    int diff_val = abs( frame1(j, i).x - frame2(j, i).x) + abs(frame1(j, i).y - frame2(j, i).y) + abs(frame1(j, i).z - frame2(j, i).z);
    if(diff_val > 20) buffer(j, i) = 20;

    if (buffer(j, i) > 0) {
        if(odd) {
            dark(j, i) = Min(frame1(j, i).x, Min(frame1(j, i).y, frame1(j, i).z));
            tran(j, i) = 255 - float((dark(j, i)) * 0.95);
            float r = float((float)tran(j, i) < 25 ? 25 : (float)tran(j, i)) / 255;

            result(j, i).x = Min(float(frame1(j, i).x - float(255 - tran(j, i)) * (1 - air_val0)) / r, 255);
            result(j, i).y = Min(float(frame1(j, i).y - float(255 - tran(j, i)) * (1 - air_val1)) / r, 255);
            result(j, i).z = Min(float(frame1(j, i).z - float(255 - tran(j, i)) * (1 - air_val2)) / r, 255);
        }
        else {
            dark(j, i) = Min(frame2(j, i).x, Min(frame2(j, i).y, frame2(j, i).z));
            tran(j, i) = 255 - float((dark(j, i)) * 0.95);
            float r = float((float)tran(j, i) < 25 ? 25 : (float)tran(j, i)) / 255;

            result(j, i).x = Min(float(frame2(j, i).x - float(255 - tran(j, i)) * (1 - air_val0)) / r, 255);
            result(j, i).y = Min(float(frame2(j, i).y - float(255 - tran(j, i)) * (1 - air_val1)) / r, 255);
            result(j, i).z = Min(float(frame2(j, i).z - float(255 - tran(j, i)) * (1 - air_val2)) / r, 255);
        }
    }
    else
        return;
}

__global__ void enhanceOpt_kernel(cuda::PtrStepSz<uchar3> frame1, cuda::PtrStepSz<uchar3> frame2, cuda::PtrStepSz<uchar3> result, cuda::PtrStepSz<uchar> dark,
                                  cuda::PtrStepSz<uchar> tran, cuda::PtrStepSz<uchar> buffer, bool odd, float k_val, float air_val0, float air_val1, float air_val2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if(buffer(j, i) > 0) buffer(j, i) --;
    int diff_val = abs(frame1(j, i).x - frame2(j, i).x) + abs(frame1(j, i).y - frame2(j, i).y) + abs(frame1(j, i).z - frame2(j, i).z);

    if(diff_val > 20) buffer(j, i) = 20;
    if (buffer(j, i) > 0) {
        if(odd) {
            dark(j, i) = Min(frame1(j, i).x, Min(frame1(j, i).y, frame1(j, i).z));
            uchar val = 255 - float((dark(j, i)) * 0.8);
            tran(j, i) = (val > 204) ? val : 204;

            float tran_scale = ((float)tran(j, i) / 255);
            result(j, i).x = Min((float)(((frame1(j, i).x) / tran_scale * k_val) * air_val0), 255);
            result(j, i).y = Min((float)(((frame1(j, i).y) / tran_scale * k_val) * air_val1), 255);
            result(j, i).z = Min((float)(((frame1(j, i).z) / tran_scale * k_val) * air_val2), 255);
        }
        else {
            dark(j, i) = Min(frame2(j, i).x, Min(frame2(j, i).y, frame2(j, i).z));
            uchar val = 255 - float((dark(j, i)) * 0.8);
            tran(j, i) = (val > 204) ? val : 204;

            float tran_scale = ((float)tran(j, i) / 255);
            result(j, i).x = Min((float)(((frame2(j, i).x) / tran_scale * k_val) * air_val0), 255);
            result(j, i).y = Min((float)(((frame2(j, i).y) / tran_scale * k_val) * air_val1), 255);
            result(j, i).z = Min((float)(((frame2(j, i).z) / tran_scale * k_val) * air_val2), 255);
        }
    }
    else
        return;
}

__global__ void colorCorrection_kernel(cuda::PtrStepSz<uchar3> src, cuda::PtrStepSz<uchar3> dst, float air_val0, float air_val1, float air_val2)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    dst(j, i).x = Min((float)(((src(j, i).x) * air_val0 )), 255);
    dst(j, i).y = Min((float)(((src(j, i).y) * air_val1 )), 255);
    dst(j, i).z = Min((float)(((src(j, i).z) * air_val2 )), 255);
}

__global__ void gammaCorrection_kernel(cuda::PtrStepSz<uchar3> src, cuda::PtrStepSz<uchar3> dst, cuda::PtrStepSz<uchar> table)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    dst(j, i).x = table(0, Min((float) src(j, i).x, 255));	// Blue Info
    dst(j, i).y = table(0, Min((float) src(j, i).y, 255));  // Green Info
    dst(j, i).z = table(0, Min((float) src(j, i).z, 255));  // Red Info
}


void getImage_WithGamma(cuda::GpuMat src, cuda::GpuMat dst, cuda::GpuMat &GTable, dim3 numBlocks, dim3 threadsPerBlock)
{
    gammaCorrection_kernel <<< numBlocks, threadsPerBlock >>> (src, dst, GTable);
}

void genCornerHomo(cuda::GpuMat &gpuH_inv, cuda::GpuMat &maps_x, cuda::GpuMat &maps_y, dim3 numBlocks, dim3 threadsPerBlock)
{
    map_kernel <<< numBlocks, threadsPerBlock >>>(gpuH_inv, maps_x, maps_y);
}

void getStitch(cuda::GpuMat &bottom_layer, cuda::GpuMat &top_layer, cuda::GpuMat &dst_layer, dim3 numBlocks, dim3 threadsPerBlock)
{
    stitch_kernel <<< numBlocks, threadsPerBlock >>>(bottom_layer, top_layer, dst_layer);
}

void getStitchOpt(cuda::GpuMat &bottom_layer, cuda::GpuMat &top_layer1, cuda::GpuMat &top_layer2, cuda::GpuMat &top_buffer, cuda::GpuMat &dst_layer, bool isOdd, dim3 numBlocks, dim3 threadsPerBlock)
{
    stitchOpt_kernel <<< numBlocks, threadsPerBlock >>>(bottom_layer, top_layer1, top_layer2, top_buffer, dst_layer, isOdd);
}

void getDarkChannel(cuda::GpuMat src, cuda::GpuMat dark_ch, dim3 numBlocks, dim3 threadsPerBlock)
{
    darkch_kernel <<< numBlocks, threadsPerBlock >>> (src, dark_ch);
}

void getTransmit(cuda::GpuMat src, cuda::GpuMat trans, dim3 numBlocks, dim3 threadsPerBlock)
{
    tran_kernel <<< numBlocks, threadsPerBlock >>> (src, trans);
}

void getDehaze(cuda::GpuMat src, cuda::GpuMat dehaze_res, cuda::GpuMat trans, float air_val[], dim3 numBlocks, dim3 threadsPerBlock)
{
    dehaze_kernal <<< numBlocks, threadsPerBlock >>> (src, dehaze_res, trans, air_val[0],air_val[1],air_val[2]);
}

void getDehaze_opt(cuda::GpuMat frame1, cuda::GpuMat frame2, cuda::GpuMat result, cuda::GpuMat darks, cuda::GpuMat trans,
                   cuda::GpuMat buffers, bool isOdd, float air_val[], dim3 numBlocks, dim3 threadsPerBlock)
{
    dehazeOpt_kernal <<< numBlocks, threadsPerBlock >>> (frame1, frame2, result, darks, trans, buffers, isOdd, air_val[0], air_val[1], air_val[2]);
}

void getLowLightTransmit(cuda::GpuMat src, cuda::GpuMat trans, dim3 numBlocks, dim3 threadsPerBlock)
{
    enhanceTrans_kernel <<< numBlocks, threadsPerBlock >>> (src, trans);
}

void getEnhancement(cuda::GpuMat src, cuda::GpuMat enhance_res, cuda::GpuMat trans, float enhace_val, float air_val[], dim3 numBlocks, dim3 threadsPerBlock)
{
    enhance_kernel <<< numBlocks, threadsPerBlock >>> (src, enhance_res, trans, enhace_val, air_val[0], air_val[1], air_val[2]);
}

void getEnhancement_opt(cuda::GpuMat frame1, cuda::GpuMat frame2, cuda::GpuMat result, cuda::GpuMat darks, cuda::GpuMat trans,
                        cuda::GpuMat buffers, bool isOdd, float enhace_val, float air_val[], dim3 numBlocks, dim3 threadsPerBlock)
{
    enhanceOpt_kernel<<< numBlocks, threadsPerBlock >>>(frame1, frame2, result, darks, trans, buffers, isOdd, enhace_val, air_val[0], air_val[1], air_val[2]);
}
